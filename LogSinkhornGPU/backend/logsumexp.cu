#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Compiling this with --use_fast_math we get some noticeable performance improvement
template <typename scalar_t>
__global__ void logsumexp(int B, int M, int N, 
    scalar_t *alpha, scalar_t *beta, scalar_t dx, scalar_t dy)
{
    // Compute the online logsumexp of a[b, i] - |x[b, i] - y[b, j]|^2, along
    // index i, for x equispaced points starting in 0 and separation dx, 
    // y equispaced point starting in 0 and separation dy.
    // alpha is input, with size (B, N)
    // beta is output, with size (B, M)

    // linear index for beta, corresponds to cartesian (b, j)
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    int b = index / M;
    int j = index % M;
    // y coordinate corresponding to beta[index]
    scalar_t y = j*dy;
    if (b >= B)
    { // take care of bigger-than-size indices
        return;
    }
    scalar_t m = -1e30f; // Initialize max for logsumexp stabilization
    // Compute max in first pass. Proved to be faster than online + update
    for (int i = 0; i < N; i++)
    {
        m = max(m, alpha[b * N + i] - (y - i*dx) * (y - i*dx));
    }
    // Compute stabilized logsumexp
    scalar_t s = 0.0f;
    for (int i = 0; i < N; i++)
    {
        s += exp(alpha[b * N + i] - (y - i*dx) * (y - i*dx) - m);
    }
    // Remove stabilization
    beta[index] = log(s) + m;
}

torch::Tensor LogSumExpCUDAKernel(
    torch::Tensor alpha, 
    int M, 
    torch::Tensor dx, 
    torch::Tensor dy) {
  
  int B = alpha.size(0);
  int N = alpha.size(1);

  // Init tensor of size (B, M)
  torch::Tensor beta = torch::empty({B, M}, alpha.options());  

  const int threads = 256;
  const int blocks = (B * M + threads - 1) / threads;
  // Dispatch dynamically as a function of alpha's type. 
  AT_DISPATCH_FLOATING_TYPES(alpha.scalar_type(), "logsumexp_cuda_kernel", ([&] {
        logsumexp<scalar_t><<<blocks, threads>>>(
            B, M, N,
            alpha.data_ptr<scalar_t>(), beta.data_ptr<scalar_t>(),
            dx.item<scalar_t>(), dy.item<scalar_t>());
    }));

  hipDeviceSynchronize();
  return beta;
}
