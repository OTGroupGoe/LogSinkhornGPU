#include "hip/hip_runtime.h"
#include "src/utils.hpp"
#include <math.h>
#include <iostream>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

////////////////////////////
// logsumexp kernel
////////////////////////////

// Compiling this with --use_fast_math we get some noticeable performance improvement
__global__ void logsumexp(int B, int M, int N, float *alpha, float *beta, float dx)
{
    // alpha is input, with size (B, N)
    // beta is output, with size (B, M)
    int index = blockIdx.x * blockDim.x + threadIdx.x; // linear index for beta, corresponds to cartesian (b, i)
    int b = index / M;
    int i = index % M;
    if (b >= B){ // take care of bigger-than-size indices
        return;
    }
    dx = dx*dx; // turn dx to dx^2 (saves multiplications below)
    float m = -1e30f; // Initialize max for logsumexp stabilization
    // Compute max in first pass. Proved to be faster than online + update
    for (int j = 0; j<N; j++)
    {
        m = max(m, alpha[b*N+j] - (i-j)*(i-j)*dx); 
    }
    // Compute stabilized logsumexp
    float s = 0.0f;
    for (int j = 0; j<N; j++)
    {
        s += exp(alpha[b*N+j] - (i-j)*(i-j)*dx - m);
    }
    // Remove stabilization
    beta[index] = log(s)+m;
}

void LogSumExpCUDAKernel(int B, int M, int N, float *alpha, float *beta, float dx)
{
  // number of elements to process is size of beta, this is, B*M
  int blockSize = 256;
  int numBlocks = (B*M + blockSize - 1) / blockSize; 
  logsumexp<<<numBlocks, blockSize>>>(B, M, N, alpha, beta, dx);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (hipSuccess != err)
    throw std::runtime_error(Formatter()
                             << "CUDA kernel failed : " << std::to_string(err));
}

//////////////////////////////////////////////////////////////////////
// inner newton method kernel for unbalanced domdec with KL divergence
//////////////////////////////////////////////////////////////////////

__global__ void inner_newton(
    int n_iter, float tol, int N, float *t, float eps, float lam,
    float *lognu, float *lognu_nJ, float *logKTu
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    float lameps = lam/eps;
    float epslam = eps/lam;
    float vmin, vmax, lse, g, g_prime; 
    float err = tol + 1.0f;

    int i = 0;
    while (i < n_iter && err > tol) {
        i++;
        // logsumexp for 2 values
        vmin = min(lognu_nJ[idx], -lameps*t[idx]);
        vmax = max(lognu_nJ[idx], -lameps*t[idx]);
        lse = vmax + log(exp(vmin - vmax) + 1.0f);

        // compute g(t[idx]) and g'(t[idx])
        g = lse - t[idx] - lognu[idx] - epslam*logKTu[idx];
        g_prime = -lameps/(1 + exp(lameps*t[idx] + lognu_nJ[idx])) - 1;
        
        // update t
        t[idx] = t[idx] - g/g_prime;
        err = g;
    }
}

void InnerNewtonCUDAKernel(
    int n_iter, float tol, int N, float *t, float eps, float lam,
    float *lognu, float *lognu_nJ, float *logKTu
) {
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    inner_newton<<<numBlocks, blockSize>>>(
        n_iter, tol, N, t, eps, lam, 
        lognu, lognu_nJ, logKTu
    );
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
        throw std::runtime_error(
            Formatter() << "CUDA kernel failed : " << std::to_string(err)
        );
}